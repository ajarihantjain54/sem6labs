#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void mulRow(int *a , int *b , int *c , int wb , int cm)
{
	int id = threadIdx.x;
	int sum = 0;
	int i = 0;
	int j = 0;
	for(i = 0;i<wb;i++)
	{
		sum = 0;
		for(j=0;j<cm;j++)
			sum += a[id*cm+j]*b[wb*j+i];
		c[id*wb+i] = sum;
	}
}

__global__ void mulCol(int *a , int *b ,int *c,int ha , int wa , int wb)
{
	int  id = threadIdx.x;
	int sum ,j,i;
	for(i = 0;i<ha;i++)
	{
		sum = 0;
		for(j = 0;j<wa;j++)
		{
			sum += a[i*wa+j]*b[j*wb+id];
		}
		c[i*wb+id] = sum;
	}
}

__global__ void mulElement(int *a ,int *b , int *c,int wa , int wb)
{
	int rid = threadIdx.x;
	int cid = threadIdx.y;
	printf("rid = %d , cid = %d \n",rid,cid);
	int sum , i;
	sum = 0;
	for(i  = 0;i<wa;i++)
	{
	sum += a[rid*wa+i]*b[i*wb+cid];
	}
	c[rid*wb+cid] = sum;
}

int main(void)
{
	int *a,*b,*t,i,j;
	int *d_a,*d_b,*d_t;
	int ha , wa;
	int hb , wb;
	printf("Enter the dimensions of first matrix \n ");
	scanf("%d %d",&ha,&wa);
	printf("Enter the dimensions of second matrix \n");
	scanf("%d %d",&hb,&wb);
	int size1 = sizeof(int)*ha*wa;
	int size2 = sizeof(int)*hb*wb;
	int size3 = sizeof(int)*ha*wb;
	a = (int*)malloc(ha*wa*sizeof(int));
	b = (int*)malloc(hb*wb*sizeof(int));
	t = (int*)malloc(ha*wb*sizeof(int));

	printf("Enter input matrix 1 : \n");
	for(i = 0;i<ha*wa;i++)
		scanf("%d",&a[i]);

	printf("Enter input matrix 2 : \n");
	for(i = 0;i<hb*wb;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMalloc((void**)&d_t,size3);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size2,hipMemcpyHostToDevice);

	printf("Enter 1 for Row \n 2 for Column \n 3 for Element \n");
	int ch;
	scanf("%d",&ch);
	if(ch == 1)
	{
	dim3 block(wb,1);
	dim3 grid(1,1);
	mulRow<<<grid,block>>>(d_a,d_b,d_t,wb,wa);
	}
	if(ch == 2)
	{
	dim3 block(ha,1);
	dim3 grid(1,1);
	mulCol<<<grid,block>>>(d_a,d_b,d_t,ha,wa,wb);
	}
	if(ch == 3)
	{
	dim3 block(ha,wb);
	dim3 grid(1,1);
	mulElement<<<grid,block>>>(d_a,d_b,d_t,wa,wb);
	}

	hipMemcpy(t,d_t,size3,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<ha;i++)
	{
		for(j = 0;j<wb;j++)
		printf("%d ",t[i*wb+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}

