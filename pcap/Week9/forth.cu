#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void complement(int* a , int* b,int n)
{
	int id = threadIdx.x;
	int m = blockDim.x;
	int j = 0;
	if(id!=0 && id!=(m-1))
	{
		for(j=1;j<n-1;j++)
		{
			int rem = 0,p=0;
			int d = a[id*m+j];
			for(p=1;d>0;p = p*10)
			{
				rem = d%2;
				if(rem == 0)
					rem =1;
				else
					rem = 0;
				b[id*m+j] += p*rem;
				d = d/2;
			}
		}
		b[id*m+0] = a[id*m+0];
		b[id*m+n-1] = a[id*m+n-1];
	}
	else
	{
		for(j=0;j<n;j++)
			b[id*m+j] = a[id*m+j];
	}

}

int main(void)
{
	int *a,*t,n,m,i,j;
	int *d_a,*d_t;
	printf("Enter the value of m and n ");
	scanf("%d",&m);
	scanf("%d",&n);
	int size = sizeof(int)*m*n;
	a = (int*)malloc(m*n*sizeof(int));
	t = (int*)malloc(m*n*sizeof(int));
	printf("Enter input matrix: \n");
	for(i = 0;i<m*n;i++)
		scanf("%d",&a[i]);
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	dim3 threadsPerBlock(m,1);
	dim3 numBlocks(1,1);
	complement<<<numBlocks,threadsPerBlock>>>(d_a,d_t,n);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<n;i++)
	{
		for(j = 0;j<n;j++)
			printf("%d ",t[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}