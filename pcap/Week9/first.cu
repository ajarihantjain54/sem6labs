#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void transpose(int *a,int *t)
{
	int v = threadIdx.y;
	int n = v*blockDim.x+threadIdx.x;
	int ta = (int)powf(a[n],v+1);
	t[n] = ta;
	printf("%d row %d element  = %d  result  \n",v,n,ta);
}


int main(void)
{
	int *a,*t,n,i,j;
	int *d_a,*d_t;
	printf("Enter the value of n: ");
	scanf("%d",&n);
	int size = sizeof(int)*n*n;
	a = (int*)malloc(n*n*sizeof(int));
	t = (int*)malloc(n*n*sizeof(int));
	printf("Enter input matrix: \n");
	for(i = 0;i<n*n;i++)
		scanf("%d",&a[i]);
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	dim3 threadsPerBlock(n,n);
	dim3 numBlocks(1,1);
	transpose<<<numBlocks,threadsPerBlock>>>(d_a,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<n;i++)
	{
		for(j = 0;j<n;j++)
			printf("%d ",t[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}