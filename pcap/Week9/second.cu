#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void addElement(int *a,int *b,int *t)
{
	int v = threadIdx.y;
	int n = v*blockDim.x+threadIdx.x;
	t[n] = a[n]+b[n];
}

__global__ void addCol(int *a , int *b , int *t)
{
	int lp =0;
	int index = threadIdx.x;
 	for(lp = 0 ;lp<blockDim.x;lp++)
 	{	
 		t[index] = a[index]+b[index];
 		index += blockDim.x;
 	}

}

__global__ void addRow(int *a , int *b , int *t)
{
 	int lp =0;
 	int index = threadIdx.x*blockDim.x;
 	for(lp = 0 ;lp<blockDim.x;lp++)
 	{	
 	t[index] = a[index] + b[index];
 	index++;
 	}

}




int main(void)
{
	int *a,*b,*t,n,i,j;
	int *d_a,*d_b,*d_t;
	printf("Enter the value of n: ");
	scanf("%d",&n);
	int size = sizeof(int)*n*n;
	a = (int*)malloc(n*n*sizeof(int));
	b = (int*)malloc(n*n*sizeof(int));
	t = (int*)malloc(n*n*sizeof(int));

	printf("Enter input matrix 1 : \n");
	for(i = 0;i<n*n;i++)
		scanf("%d",&a[i]);

	printf("Enter input matrix 2 : \n");
	for(i = 0;i<n*n;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	printf("Enter 1 for Row \n 2 for Column \n 3 for Element \n");
	int ch;
	scanf("%d",&ch);
	if(ch == 1)
	{
	dim3 block(n,1);
	dim3 grid(1,1);
	addRow<<<grid,block>>>(d_a,d_b,d_t);
	}
	if(ch == 2)
	{
	dim3 block(n,1);
	dim3 grid(1,1);
	addCol<<<grid,block>>>(d_a,d_b,d_t);
	}
	if(ch == 3)
	{
	dim3 block(n,n);
	dim3 grid(1,1);
	addElement<<<grid,block>>>(d_a,d_b,d_t);
	}

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<n;i++)
	{
		for(j = 0;j<n;j++)
		printf("%d ",t[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}