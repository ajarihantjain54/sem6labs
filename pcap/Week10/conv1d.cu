#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void convolution_1D(float *N , float *M , float *P , int Mask_width,int width)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float pvalue = 0.0;
	int N_start_point = i - ((Mask_width)/2);
	for(int j =0;j<Mask_width;j++)
	{
		if(((N_start_point+j)>=0)&& ((N_start_point + j)<width))
		{
			pvalue  += N[N_start_point + j] * M[j];
		}
	}
	P[i] = pvalue;
}

int main()
{
	float *a , *m , *t;
	float *d_a, *d_m, *d_t;
	int width , mask_width;
	printf("Enter the size of array \n");
	scanf("%d",&width);
	a = (float*)malloc(sizeof(float)*width);
	t = (float*)malloc(sizeof(float)*width);
	printf("Enter the array \n");
	int i = 0;
	for(i = 0;i<width;i++)
	{
		scanf("%f",&a[i]);
	}
	printf("Enter the size of mask \n");
	scanf("%d",&mask_width);
	m = (float*)malloc(sizeof(float)*mask_width);
	printf("Enter the mask \n");
	for(i = 0;i<mask_width;i++)
	{
		scanf("%f",&m[i]);
	}
	int size1 = sizeof(float)*width;
	int size2 = sizeof(float)*mask_width;
	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_m,size2);
	hipMalloc((void**)&d_t,size1);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_m,m,size2,hipMemcpyHostToDevice);
	dim3 dimGrid((width-1)/mask_width + 1,1,1);
	dim3 dimBlock(mask_width,1,1);
	convolution_1D<<<dimGrid,dimBlock>>>(d_a,d_m,d_t,mask_width,width);
	hipMemcpy(t,d_t,size1,hipMemcpyDeviceToHost);
	printf("The result array is \n");
	for(int i = 0;i<width;i++)
	{
		printf("%f ",t[i]);
	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_m);
	hipFree(d_t);


}