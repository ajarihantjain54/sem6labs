#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int getTid()
{
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int tPB = blockDim.x * blockDim.y ;
	int fin = bid*tPB+tid;
}

__global__ void mulElement(int *a ,int *b , int *c , int ha , int wb,int wa)
{
	int th = getTid();
	if(th<(ha*wb))
	{
		int row = th/wb;
		int col = th%wb;
		int i = 0 , sum = 0;
		for(i = 0;i<wa;i++)
		{
			sum += a[row*wa+i]*b[wb*i+col];
		}
		c[th] = sum;
	}
	
}

int main(void)
{
	int *a,*b,*t,i,j;
	int *d_a,*d_b,*d_t;
	int ha , wa;
	int hb , wb;
	printf("Enter the dimensions of first matrix \n ");
	scanf("%d %d",&ha,&wa);
	printf("Enter the dimensions of second matrix \n");
	scanf("%d %d",&hb,&wb);
	int size1 = sizeof(int)*ha*wa;
	int size2 = sizeof(int)*hb*wb;
	int size3 = sizeof(int)*ha*wb;
	a = (int*)malloc(ha*wa*sizeof(int));
	b = (int*)malloc(hb*wb*sizeof(int));
	t = (int*)malloc(ha*wb*sizeof(int));

	printf("Enter input matrix 1 : \n");
	for(i = 0;i<ha*wa;i++)
		scanf("%d",&a[i]);

	printf("Enter input matrix 2 : \n");
	for(i = 0;i<hb*wb;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMalloc((void**)&d_t,size3);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size2,hipMemcpyHostToDevice);
	int gx,gy,bx,by;
	printf("Enter the dimension of the grid \n");
	scanf("%d %d",&gx,&gy);
	bx = ceil((double)ha/gx);
	by = ceil((double)wb/gy);
	printf("The dimensions of block are : \n %d %d \n",bx,by);
	dim3 grid(gx,gy);
	dim3 block(bx,by);
	mulElement<<<grid,block>>>(d_a,d_b,d_t,ha,wb,wa);
	hipMemcpy(t,d_t,size3,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<ha;i++)
	{
		for(j = 0;j<wb;j++)
		printf("%d ",t[i*wb+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}