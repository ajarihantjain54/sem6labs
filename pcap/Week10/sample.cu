#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int getTid()
{
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int tPB = blockDim.x * blockDim.y ;
	int fin = bid*tPB+tid;
	return fin;
}

__global__ void mulElement(int *a ,int *b , int *c , int ha ,int wa)
{
	int th = getTid();
	if(th<(ha*wa))
	{
		c[th] = a[th]+b[th];
	}
	
}

int main(void)
{
	int *a,*b,*t,i,j;
	int *d_a,*d_b,*d_t;
	int ha , wa;
	printf("Enter the dimensions of first matrix \n ");
	scanf("%d %d",&ha,&wa);
	int size1 = sizeof(int)*ha*wa;
	a = (int*)malloc(size1);
	b = (int*)malloc(size1);
	t = (int*)malloc(size1);

	printf("Enter input matrix 1 : \n");
	for(i = 0;i<ha*wa;i++)
		scanf("%d",&a[i]);

	printf("Enter input matrix 2 : \n");
	for(i = 0;i<ha*wa;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size1);
	hipMalloc((void**)&d_t,size1);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size1,hipMemcpyHostToDevice);
	int gx,gy,bx,by;
	printf("Enter the dimension of the grid \n");
	scanf("%d %d",&gx,&gy);
	bx = ceil((double)ha/gx);
	by = ceil((double)wa/gy);
	printf("The dimensions of block are : \n %d %d \n",bx,by);
	dim3 grid(gx,gy);
	dim3 block(bx,by);
	mulElement<<<grid,block>>>(d_a,d_b,d_t,ha,wa);
	hipMemcpy(t,d_t,size1,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<ha;i++)
	{
		for(j = 0;j<wa;j++)
		printf("%d ",t[i*wa+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}