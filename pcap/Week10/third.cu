#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>



int main(void)
{
	int *a,*b,*t,i,j;
	int *d_a,*d_b,*d_t;
	int ha , wa;
	int hb , wb;
	printf("Enter the dimensions of first matrix \n ");
	scanf("%d %d",&ha,&wa);
	printf("Enter the dimensions of second matrix \n");
	scanf("%d %d",&hb,&wb);
	int size1 = sizeof(int)*ha*wa;
	int size2 = sizeof(int)*hb*wb;
	int size3 = sizeof(int)*ha*wb;
	a = (int*)malloc(ha*wa*sizeof(int));
	b = (int*)malloc(hb*wb*sizeof(int));
	t = (int*)malloc(ha*wb*sizeof(int));

	printf("Enter sparse matrix  : \n");
	for(i = 0;i<ha*wa;i++)
		scanf("%d",&a[i]);

	printf("Enter input matrix 2 : \n");
	for(i = 0;i<hb*wb;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMalloc((void**)&d_t,size3);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size2,hipMemcpyHostToDevice);
	
	mulElement<<<1,ha>>>(d_a,d_b,d_t,ha,wb,wa);
	hipMemcpy(t,d_t,size3,hipMemcpyDeviceToHost);
	printf("Result vector is :\n");
	for(i = 0;i<ha;i++)
	{
		for(j = 0;j<wb;j++)
		printf("%d ",t[i*wb+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}