#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>



__global__ void add(float *a , float *b)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	b[id] = sinf(a[id]);
}

int main(void)
{
	float *a , *b ;
	float *d_a , *d_b ;
	printf("Enter the value of N \n");
	int n;
	int i;
	scanf("%d",&n);
	a = (float*)malloc(sizeof(float)*n);
	b = (float*)malloc(sizeof(float)*n);
	printf("Enter the values for 1st Array \n");
	for( i = 0;i<n;i++)
		scanf("%f",&a[i]);
	int size = sizeof(float)*n;

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<1,n>>>(d_a,d_b);

	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
	printf("Result \n");
	for( i = 0;i<n;i++)
		printf("%f \t",b[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
