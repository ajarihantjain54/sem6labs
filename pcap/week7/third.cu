
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>



__global__ void add(int* a , int *b,int *c)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	b[id] = (a[id] * (*c)) + b[id];
	//printf("A = %d \t B = %d \t C = %d \n",a[id],b[id],*c);
}

int main(void)
{
	int *a , *b , c;
	int *d_a , *d_b ,*d_c;
	printf("Enter the value of N \n");
	int n;
	int i;
	scanf("%d",&n);
	a = (int*)malloc(sizeof(int)*n);
	b = (int*)malloc(sizeof(int)*n);
	printf("Enter the value of alpha \n");
		scanf("%d",&c);
	printf("Enter the values for 1st Array \n");
	for( i = 0;i<n;i++)
		scanf("%d",&a[i]);
	printf("Enter the values for 2nd Array \n");
	for(i = 0;i<n;i++)
		scanf("%d",&b[i]);
	int size = sizeof(int)*n;

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,sizeof(int));
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	hipMemcpy(d_c,&c,sizeof(int),hipMemcpyHostToDevice);
	add<<<1,n>>>(d_a,d_b,d_c);

	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
	printf("Result \n");
	for( i = 0;i<n;i++)
		printf("%d \t",b[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
