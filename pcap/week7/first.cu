
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>



__global__ void add(int* a , int *b,int *c)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	c[id] = a[id] + b[id];
	printf("A = %d \t B = %d \t C = %d \n",a[id],b[id],c[id]);
}

int main(void)
{
	int *a , *b ,*c;
	int *d_a , *d_b ,*d_c;
	printf("Enter the value of N \n");
	int n;
	int i;
	scanf("%d",&n);
	printf("Enter your choice \n");
	int ch = 0;
	scanf("%d",&ch);
	a = (int*)malloc(sizeof(int)*n);
	b = (int*)malloc(sizeof(int)*n);
	c = (int*)malloc(sizeof(int)*n);
	printf("Enter the values for 1st Array \n");
	for( i = 0;i<n;i++)
		scanf("%d",&a[i]);
	printf("Enter the values for 2nd Array \n");
	for(i = 0;i<n;i++)
		scanf("%d",&b[i]);
	int size = sizeof(int)*n;

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	if(ch == 1)
		add<<<n,1>>>(d_a,d_b,d_c);
	else if(ch==2)
		add<<<1,n>>>(d_a,d_b,d_c);
	else if(ch == 3)
		add<<<n,256>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("Result \n");
	for( i = 0;i<n;i++)
		printf("%d \t",c[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
